
#include <hip/hip_runtime.h>
extern "C"

#define max(a, b) (a > b)? a: b
#define min(a, b) (a > b)? b: a


__global__ void MergerHelper1(double *seedHBMax, double *seedHBMin, double *mergingHBMaxes, double *mergingHBMins, double *combinedMax, double *combinedMin, double *opClassPnts, int *toBeDeleted, int numDims, int numMergingHBs, int cases)
{
    int n = blockIdx.x * blockDim.x + threadIdx.x;
    if (n < numMergingHBs)
    {
        int offset = n * numDims;
        for (int i = 0; i < numDims; i++)
        {
            combinedMax[i+offset] = max(seedHBMax[i], mergingHBMaxes[i+offset]);
            combinedMin[i+offset] = min(seedHBMin[i], mergingHBMins[i+offset]);
        }

        // 1 = do merge, 0 = do not merge
        int merge = 1;
        for (int i = 0; i < cases; i += numDims)
        {
            bool withinSpace = true;
            for (int j = 0; j < numDims; j++)
            {
                if (!(opClassPnts[i+j] <= combinedMax[j+offset] && opClassPnts[i+j] >= combinedMin[j+offset]))
                {
                    withinSpace = false;
                    break;
                }
            }

            if (withinSpace)
            {
                merge = 0;
                break;
            }
        }

        toBeDeleted[n] = merge;
    }
}
